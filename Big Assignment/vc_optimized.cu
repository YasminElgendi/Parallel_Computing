#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <queue>
#include "graph.h"
#include "timer.h"

#define LOCAL_QUEUE_SIZE 2048

// Objective of BFS: To find the shortest path from a source vertex to all other vertices in an unweighted graph
// output file: the line number corresponds to the vertex number and the value corresponds to the level of the vertex from the source vertex

/*
 Vertex Centric Paralleisation of BFS:
    1. Each vertex is assigned a thread
    2. Each thread processes the vertex and its neighbours

    A vertex-centric parallel implementation assigns threads to vertices and has each thread perform an operation on its vertex, which usually involves
    iterating over the neighbors of that vertex

    There are 2 implementations:
    1. Focuses on the outgoing edges of the vertex (top-down approach)
        Since the CSR format stores the outgoing edges of a vertex, we eill use it for this approach
    2. Focuses on the incoming edges of the vertex

    The function will be called for each level in the graph

    The optimized version of the vertex centric push BFS will use the following:
    1. Frontiers: We will make use of frontiers to reduce redundant work:
        - The redundency from the previous implementation results from that each thread will loop over all the neighbours of the vertex to find them
    2. Privatization: shared memory will be used to store the frontier => to deal with atomic operations
        - since the frontier is shared with all threads in the same block we will use shared memory to store the frontier
        - All threads are atomically incrementing the same global counter to insert elements in the queue => high latency (global memory access) and serialization (contention)
        - Each block will commit to a private queue and then commit entrie to global queue
    3. Minimized Launch Overhead
        - The kernel is only called when the number of vertices of two consecutive queues combined is less than the number of threads in a single block
        - This is to minimize the overhead of the kernel launch
        - This will create a local level array for the block and synchronize at the end
*/

// For later use: hipDeviceSynchronize() => Waits for all kernels in all streams on a CUDA device to complete.

__global__ void vertex_centric_optimized_bfs(unsigned int *srcPtrs, unsigned int *dst, unsigned int *level, unsigned int *currentQueue,
                                             unsigned int *previousQueue, int currentLevel, unsigned int *numberOfCurrentQueue, int numberOfPreviousQueue,
                                             int vertices, int edges, unsigned int *numberOfLevels)
{
    // define the shared memory for the frontier
    __shared__ unsigned int sharedCurrentQueue[LOCAL_QUEUE_SIZE];
    __shared__ unsigned int sharedNumCurrentFrontier;
    __shared__ unsigned int sharedNumLevels;

    // only need a single thread to initialize the shared memory not all but make sure all threads are synchronized
    if (threadIdx.x == 0)
    {
        // printf("Regular kernel\n");
        sharedNumLevels = 1;
        sharedNumCurrentFrontier = 0;
    }

    __syncthreads();

    // Perform BFS on local queue
    // each thread is assigned a vertex
    // since this is considered a 1-D array we will use the the x index to get the vertex for each thread
    // following the basis of a vector addition
    unsigned int vertexIndex = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary conditions => check if the vertex within the previous queue
    if (vertexIndex < numberOfPreviousQueue)
    {
        unsigned int vertex = previousQueue[vertexIndex]; // get the vertex from the previous queue

        unsigned int start = srcPtrs[vertex];   // get the start index of the neighbours of the vertex
        unsigned int end = srcPtrs[vertex + 1]; // get the end index of the neighbours of the vertex

        // iterate over the neighbours of the vertex
        for (unsigned int i = start; i < end; i++)
        {
            unsigned int neighbour = dst[i];
            // the atomic compare and swap doesnt change since the level array is still global and shared across all threads
            // what happens if multiple threads in the previous queue have the same neighbours => race condition (can be added to the current queue multiple times)
            if (atomicCAS(&level[neighbour], UINT_MAX, currentLevel) == UINT_MAX)
            {
                // Add to the local queue
                unsigned int sharedIndex = atomicAdd(&sharedNumCurrentFrontier, 1); // get the index of the local current queue

                // Check if the shared memory is full
                if (sharedIndex < LOCAL_QUEUE_SIZE)
                {
                    sharedCurrentQueue[sharedIndex] = neighbour;
                }
                else
                {
                    // dealing with overflow
                    // since the shared memory is of limited size we will add the neighbour to the global queue if the shared memory is full

                    sharedNumCurrentFrontier = LOCAL_QUEUE_SIZE;             // set the number of the current queue to the max size
                    unsigned int index = atomicAdd(numberOfCurrentQueue, 1); // add the neighbour to the current queue
                    currentQueue[index] = neighbour;
                }
            }
        }
    }
    __syncthreads(); // wait for all threads before copying the shared memory to the global memory

    // copy the shared memory to the global memory
    __shared__ unsigned int sharedStartIndex; // get the start index of the current queue in the global memory

    if (threadIdx.x == 0)
    {
        sharedStartIndex = atomicAdd(numberOfCurrentQueue, sharedNumCurrentFrontier);

        // copy the number of levels to the global memory
        *numberOfLevels = sharedNumLevels;
    }

    __syncthreads();

    for (unsigned int i = threadIdx.x; i < sharedNumCurrentFrontier; i += blockDim.x) // memory coalescing achieved
    {
        currentQueue[sharedStartIndex + i] = sharedCurrentQueue[i];
    }
}

// The kernel is only called when the number of vertices of two consecutive queues combined is less than the number of threads in a single block
// This is to minimize the overhead of the kernel launch
// This will create a local level array for the block and synchronize at the end
__global__ void minimize_overhead_kernel(unsigned int *srcPtrs, unsigned int *dst, unsigned int *level, unsigned int *currentQueue,
                                         unsigned int *previousQueue, int currentLevel, unsigned int *numberOfCurrentQueue, int numberOfPreviousQueue,
                                         int vertices, int edges, unsigned int *numberOfLevels)
{
    __shared__ unsigned int sharedPreviuosQueue[LOCAL_QUEUE_SIZE];
    __shared__ unsigned int sharedCurrentQueue[LOCAL_QUEUE_SIZE];
    __shared__ unsigned int sharedNumPreviousQueue;
    __shared__ unsigned int sharedNumCurrentQueue;
    __shared__ unsigned int sharedNumLevels;

    // Since we have a single block the vertex index in the queue is the thread index
    int threadIndex = threadIdx.x;

    // only need a single thread to initialize the shared memory not all but make sure all threads are synchronized
    if (threadIndex == 0)
    {
        sharedNumPreviousQueue = numberOfPreviousQueue;
        sharedNumCurrentQueue = 0;
        sharedNumLevels = 0;
    }
    __syncthreads();

    // Copy the previous queue to the shared memory
    for (int i = threadIndex; i < sharedNumPreviousQueue; i += blockDim.x)
    {
        sharedPreviuosQueue[i] = previousQueue[i];
    }
    __syncthreads();

    // While the number of vertices in the previous queue is greater than 0 and the number of vertices in the current queue is less than the block size
    // If the number of vertices in the current queue is greates than the block size then we will add the vertices to the global queue


    while (sharedNumPreviousQueue > 0 && sharedNumCurrentQueue <= blockDim.x)
    {
        // Reset the number of vertices in the current queue
        if (threadIndex == 0)
        {
            sharedNumCurrentQueue = 0;
        }
        __syncthreads();

        // ... 

        // Process the vertices in the queue
        for (unsigned int i = threadIndex; i < sharedNumPreviousQueue; i += blockDim.x)
        {
            unsigned int vertex = sharedPreviuosQueue[i];
            unsigned int start = srcPtrs[vertex];
            unsigned int end = srcPtrs[vertex + 1];

            for (unsigned int j = start; j < end; j++)
            {
                unsigned int neighbour = dst[j];
                if (atomicCAS(&level[neighbour], UINT_MAX, currentLevel + sharedNumLevels) == UINT_MAX)
                {
                    unsigned int sharedIndex = atomicAdd(&sharedNumCurrentQueue, 1);
                    if (sharedIndex < LOCAL_QUEUE_SIZE)
                    {
                        sharedCurrentQueue[sharedIndex] = neighbour;
                    }
                    else
                    {
                        sharedNumCurrentQueue = LOCAL_QUEUE_SIZE;
                        unsigned int index = atomicAdd(numberOfCurrentQueue, 1);
                        currentQueue[index] = neighbour;
                    }
                }
            }
        }
        __syncthreads();

        if (threadIndex == 0)
        {
            // Copy the current queue to the previous queue
            // The previous queue becomes the current queue for the next iteration
            sharedNumPreviousQueue = sharedNumCurrentQueue; 

            // Increment the number of levels to indicate a level has been processed
            atomicAdd(&sharedNumLevels, 1);
        }
        __syncthreads();

        // Each thread copies an element from the current queue to the previous queue
        for (int i = threadIndex; i < sharedNumCurrentQueue; i += blockDim.x)
        {
            sharedPreviuosQueue[i] = sharedCurrentQueue[i];
        }
        __syncthreads();


        // if (sharedNumPreviousQueue > LOCAL_QUEUE_SIZE)
        //     break;
    }

    // copy the shared memory to the global memory
    __shared__ unsigned int sharedStartIndex; // get the start index of the current queue in the global memory

    if (threadIdx.x == 0)
    {
        sharedStartIndex = atomicAdd(numberOfCurrentQueue, sharedNumCurrentQueue);

        // copy the number of levels to the global memory
        *numberOfLevels = sharedNumLevels;
    }

    __syncthreads();

    for (unsigned int i = threadIdx.x; i < sharedNumCurrentQueue; i += blockDim.x) // memory coalescing achieved
    {
        currentQueue[sharedStartIndex + i] = sharedCurrentQueue[i];
    }
}

__host__ void bfs_cpu(unsigned int *srcPtrs, unsigned int *dst, unsigned int *level, int srcVertex)
{
    // this function will be used to compare the results of the GPU with the CPU
    // define a queue to store the vertices
    std::queue<int> verticesQueue;

    // vector<bool> visited(vertices, false);

    // mark the current node as visited and enqueue it
    level[srcVertex] = 0;
    verticesQueue.push(srcVertex);

    while (!verticesQueue.empty())
    {
        // dequeue a vertex from queue and print it
        int currentVertex = verticesQueue.front();
        verticesQueue.pop();

        int start = srcPtrs[currentVertex];
        int end = srcPtrs[currentVertex + 1];

        // get all adjacent vertices of the dequeued vertex
        // if an adjacent has not been visited, then mark it visited and enqueue it
        for (int i = start; i < end; i++)
        {
            int neighbour = dst[i];
            if (level[neighbour] == UINT_MAX)
            {
                verticesQueue.push(neighbour);
                level[neighbour] = level[currentVertex] + 1;
            }
        }
    }
}

int main(char argc, char *argv[])
{

    if (argc < 4)
    {
        printf("Please provide the paths of the input and output files and the source vertex\n");
        return 1;
    }

    FILE *inputFile;
    FILE *outputFile;
    int srcVertex = atoi(argv[3]);
    Timer timer;

    inputFile = fopen(argv[1], "r");
    outputFile = fopen(argv[2], "w");

    if (!inputFile || !outputFile)
    {
        printf("Please provide the correct path of both files");
        return 1;
    }

    // 1. Allocate host memory for the graph
    int vertices, edges;

    // Read the number of vertices and edges
    fscanf(inputFile, "%d %d", &vertices, &edges);

    unsigned int *srcPtrs = (unsigned int *)malloc((vertices + 1) * sizeof(unsigned int)); // allocate with the actual number of source vertices => directed graph
    unsigned int *dst = (unsigned int *)malloc(edges * sizeof(unsigned int));
    unsigned int *level = (unsigned int *)malloc(vertices * sizeof(unsigned int));
    unsigned int *levelCPU = (unsigned int *)malloc(vertices * sizeof(unsigned int));

    // Initialize the level of each vertex to -1
    // and the source vertex to 0
    for (int i = 0; i < vertices; i++)
    {
        if (i == srcVertex)
        {
            level[i] = 0;
            levelCPU[i] = 0;
        }
        else
        {
            level[i] = UINT_MAX;
            levelCPU[i] = UINT_MAX;
        }
    }

    // Construct the graph using the CSR representation
    CSRGraph(inputFile, srcPtrs, dst, edges);

    // Run the CPU BFS
    timer.start();
    bfs_cpu(srcPtrs, dst, levelCPU, srcVertex);
    timer.stop();

    // 7. Write the result to the output file
    FILE *cpuOutputFile = fopen("output/cpu_output.txt", "w");
    for (int i = 0; i < vertices; i++)
    {
        fprintf(cpuOutputFile, "%d %d\n", i, levelCPU[i]);
    }

    double cpuTime = timer.elapsed();
    printf("\033[0;33m"); // set color to yellow
    printf("CPU Time: %f ms\n", cpuTime);
    printf("\033[0m"); // reset color

    // 2. Allocate device memory for the graph
    timer.start();
    unsigned int *deviceSrc;
    unsigned int *deviceDst;
    unsigned int *deviceLevel;
    unsigned int *deviceTemp1;           // used to swap the current and previous queues
    unsigned int *deviceTemp2;           // used to swap the current and previous queues
    unsigned int *deviceNumCurrentQueue; // the number of vertices in the queue of the current level
    unsigned int *deviceNumLevels;       // the number of levels updated in the kernel

    hipMalloc((void **)&deviceSrc, (vertices + 1) * sizeof(unsigned int));
    hipMalloc((void **)&deviceDst, edges * sizeof(unsigned int));
    hipMalloc((void **)&deviceLevel, vertices * sizeof(unsigned int));
    hipMalloc((void **)&deviceTemp1, vertices * sizeof(unsigned int));
    hipMalloc((void **)&deviceTemp2, vertices * sizeof(unsigned int));
    hipMalloc((void **)&deviceNumCurrentQueue, sizeof(unsigned int));
    hipMalloc((void **)&deviceNumLevels, sizeof(unsigned int));
    hipDeviceSynchronize();

    unsigned int *devicePreviosQueue = deviceTemp1;
    unsigned int *deviceCurrentQueue = deviceTemp2;

    // printf("\nDevice memory allocated successfully\n");
    timer.stop();
    double allocationTime = timer.elapsed();
    printf("Time taken to allocate device memory: %f ms\n", allocationTime);

    // 3. Copy memory to the device
    timer.start();
    hipMemcpy(deviceSrc, srcPtrs, (vertices + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(deviceDst, dst, edges * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(deviceLevel, level, vertices * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(deviceCurrentQueue, &srcVertex, sizeof(unsigned int), hipMemcpyHostToDevice); // the initial queue will have the source vertex only
    hipDeviceSynchronize();
    timer.stop();

    // printf("Copied to GPU successfully\n");
    double copyingTime = timer.elapsed();

    printf("Time taken to copy memory to the device: %f ms\n", copyingTime);

    // 4. Set the number of threads and blocks
    int threadsPerBlock = 256;
    int blocksPerGrid;
    int numberOfPreviousQueue = 1;

    int currentLevel = 1; // we start from level 1 since we already set the level of the source vertex to 0
    int numberOfLevels = 0;

    // 5. Launch the kernel
    timer.start();

    // Here the stopping condition changed to check the number of vertices in the previous queue
    // If the number of vertices in the previous queue is 0 then no thread has added any vertices to the current queue
    // This means that the max depth has been reached
    while (numberOfPreviousQueue > 0)
    {
        hipMemset(deviceNumCurrentQueue, 0, sizeof(unsigned int));                      // reset the number of vertices in the current queue


        hipMemset(deviceNumLevels, 0, sizeof(unsigned int));  // reset the number of vertices in the current queue


        blocksPerGrid = (numberOfPreviousQueue + threadsPerBlock - 1) / threadsPerBlock; // calculate the number of blocks needed for the current queue

        if (numberOfPreviousQueue <= threadsPerBlock)
        {
            minimize_overhead_kernel<<<1, threadsPerBlock>>>(deviceSrc, deviceDst, deviceLevel, deviceCurrentQueue, devicePreviosQueue,
                                                             currentLevel, deviceNumCurrentQueue, numberOfPreviousQueue, vertices, edges, deviceNumLevels);
        }
        else
        {
            vertex_centric_optimized_bfs<<<blocksPerGrid, threadsPerBlock>>>(deviceSrc, deviceDst, deviceLevel, deviceCurrentQueue, devicePreviosQueue,
                                                                             currentLevel, deviceNumCurrentQueue, numberOfPreviousQueue, vertices, edges, deviceNumLevels);
        }

        // Copy the number of the current queue of the device to the number of previous queue of the host
        hipMemcpy(&numberOfPreviousQueue, deviceNumCurrentQueue, sizeof(unsigned int), hipMemcpyDeviceToHost);

        // Swap the current and previous queues
        unsigned int *temp = devicePreviosQueue;
        devicePreviosQueue = deviceCurrentQueue;
        deviceCurrentQueue = temp;

        // Copy the number of levels to the host
        hipMemcpy(&numberOfLevels, deviceNumLevels, sizeof(unsigned int), hipMemcpyDeviceToHost);

        currentLevel += numberOfLevels;

    }

    hipDeviceSynchronize(); // wait for all kernels to finish so that the level array is updated
    timer.stop();
    double kernelTime = timer.elapsed();
    printf("\033[0;34m"); // set colour to blue
    printf("Kernel Time: %f ms\n", kernelTime);
    printf("\033[0m"); // reset color

    // check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        // in red
        printf("\033[0;31m");
        printf("CUDA error: %s\n", hipGetErrorString(error));
        // reset color
        printf("\033[0m");
    }

    // 6. Copy the result back to the host
    hipMemcpy(level, deviceLevel, vertices * sizeof(int), hipMemcpyDeviceToHost);

    // 7. Write the result to the output file
    for (int i = 0; i < vertices; i++)
    {
        fprintf(outputFile, "%d %d\n", i, level[i]);
    }

    // Close files
    fclose(inputFile);
    fclose(outputFile);

    // Free host memory
    free(srcPtrs);
    free(dst);
    free(level);
    free(levelCPU);

    // Free device memory
    timer.start();
    hipFree(deviceSrc);
    hipFree(deviceDst);
    hipFree(deviceLevel);
    hipFree(devicePreviosQueue);
    hipFree(deviceCurrentQueue);
    hipFree(deviceNumCurrentQueue);
    hipFree(deviceNumLevels);
    timer.stop();

    double deallocationTime = timer.elapsed();

    printf("Time taken to deallocate device memory: %f ms\n", deallocationTime);

    double totalGPUTime = allocationTime + copyingTime + kernelTime + deallocationTime;

    printf("\033[0;32m"); // set color to green
    printf("Total GPU time: %f ms\n", totalGPUTime);
    printf("\033[0m"); // reset color

    return 0;
}