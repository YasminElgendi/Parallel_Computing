#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include "graph.h"
#include "timer.h"

// Objective of BFS: To find the shortest path from a source vertex to all other vertices in an unweighted graph
// output file: the line number corresponds to the vertex number and the value corresponds to the level of the vertex from the source vertex

/*
 Vertex Centric Paralleisation of BFS:
    1. Each vertex is assigned a thread
    2. Each thread processes the vertex and its neighbours

    A vertex-centric parallel implementation assigns threads to vertices and has each thread perform an operation on its vertex, which usually involves
    iterating over the neighbors of that vertex

    There are 2 implementations:
    1. Focuses on the outgoing edges of the vertex (top-down approach)
        Since the CSR format stores the outgoing edges of a vertex, we eill use it for this approach
    2. Focuses on the incoming edges of the vertex

    The function will be called for each level in the graph

*/

// For later use: hipDeviceSynchronize() => Waits for all kernels in all streams on a CUDA device to complete.

__global__ void vertex_centric_push_bfs(unsigned int *srcPtrs, unsigned int *dst, unsigned int *level, int currentLevel, int vertices, int edges, unsigned int *vertexVisited)
{
    // each thread is assigned a vertex
    // since this is considered a 1-D array we will use the the x index to get the vertex for each thread
    // following the basis of a vector addition
    unsigned int vertex = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary conditions
    // check if the vertex is inside the graph
    if (vertex < vertices)
    {
        if (level[vertex] == currentLevel - 1) // then loop on all neighbours of the vertex
        {
            // get the starting and ending index of the edges of the vertex
            // the srcPtrs contain the starting index of the edges of the vertex for each row (vertex)
            unsigned int start = srcPtrs[vertex];   // this marks the start of the edges of the vertex
            unsigned int end = srcPtrs[vertex + 1]; // this marks the start of the edges of the next vertex => the end of the edges of the current vertex

            // iterate over the neighbours of the vertex
            for (unsigned int i = start; i < end; i++)
            {
                unsigned int neighbour = dst[i]; // get the neighbour of the vertex

                if (level[neighbour] == UINT_MAX) // check if the neighbour has not been visited
                {
                    // if the neighbour has not been visited then set the level of the neighbour to the current level
                    // and mark the vertex as visited
                    level[neighbour] = currentLevel;
                    *vertexVisited = 1;
                }
            }
        }
    }
}

int main(char argc, char *argv[])
{

    if (argc < 4)
    {
        printf("Please provide the paths of the input and output files and the source vertex\n");
        return 1;
    }

    FILE *inputFile;
    FILE *outputFile;
    int srcVertex = atoi(argv[3]);
    Timer timer;

    inputFile = fopen(argv[1], "r");
    outputFile = fopen(argv[2], "w");

    if (!inputFile || !outputFile)
    {
        printf("Please provide the correct path of both files");
        return 1;
    }

    // 1. Allocate host memory for the graph
    int vertices, edges;

    // Read the number of vertices and edges
    fscanf(inputFile, "%d %d", &vertices, &edges);

    unsigned int *srcPtrs = (unsigned int *)malloc((vertices + 1) * sizeof(unsigned int)); // allocate with the actual number of source vertices => directed graph
    unsigned int *dst = (unsigned int *)malloc(edges * sizeof(unsigned int));
    unsigned int *level = (unsigned int *)malloc(vertices * sizeof(unsigned int));
    unsigned int *srcNames = (unsigned int *)malloc((vertices + 1) * sizeof(unsigned int));

    // Initialize the level of each vertex to -1
    // and the source vertex to 0
    for (int i = 0; i < vertices; i++)
    {
        if (i == srcVertex)
            level[i] = 0;
        else
            level[i] = UINT_MAX;
    }

    // Construct the graph using the CSR representation
    CSRGraph(inputFile, srcPtrs, dst, edges);

    // 2. Allocate device memory for the graph
    timer.start();
    unsigned int *deviceSrc;
    unsigned int *deviceDst;
    unsigned int *deviceLevel;
    unsigned int *deviceVertexVisited;

    hipMalloc((void **)&deviceSrc, (vertices + 1) * sizeof(unsigned int));
    hipMalloc((void **)&deviceDst, edges * sizeof(unsigned int));
    hipMalloc((void **)&deviceLevel, vertices * sizeof(unsigned int));
    hipMalloc((void **)&deviceVertexVisited, sizeof(unsigned int));

    // printf("\nDevice memory allocated successfully\n");
    timer.stop();
    double allocationTime = timer.elapsed();
    printf("Time taken to allocate device memory: %f ms\n", allocationTime);

    // 3. Copy memory to the device
    timer.start();
    hipMemcpy(deviceSrc, srcPtrs, (vertices + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceDst, dst, edges * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceLevel, level, vertices * sizeof(int), hipMemcpyHostToDevice);
    timer.stop();

    // printf("Copied to GPU successfully\n");
    double copyingTime = timer.elapsed();

    printf("Time taken to copy memory to the device: %f ms\n", copyingTime);

    // 4. Set the number of threads and blocks
    int threadsPerBlock = 256;
    int blocksPerGrid = (vertices + threadsPerBlock - 1) / threadsPerBlock;
    unsigned int vertexVisited = 1;

    int currentLevel = 1; // we start from level 1 since we already set the level of the source vertex to 0

    // printf("Launching Kernel\n");
    // 5. Launch the kernel
    timer.start();

    while (vertexVisited)
    {
        vertexVisited = 0; // reset the vertexVisited to 0 for each level

        // copy the vertexVisited to the device before the launch of each kernel
        hipMemcpy(deviceVertexVisited, &vertexVisited, sizeof(unsigned int), hipMemcpyHostToDevice); 

        // kernel processes each level
        // the kernel will be called for each level in the graph
        // global synchronisation across different levels is required
        vertex_centric_push_bfs<<<blocksPerGrid, threadsPerBlock>>>(deviceSrc, deviceDst, deviceLevel, currentLevel, vertices, edges, deviceVertexVisited);

        // copy the vertexVisited back to the host after the kernel has finished to check whether any vertex has been newly visited 
        // if not the max depth reached then all vertices in the current level have been visited
        hipMemcpy(&vertexVisited, deviceVertexVisited, sizeof(unsigned int), hipMemcpyDeviceToHost); 

        currentLevel++; // increment the current level for the next iteration
    }

    hipDeviceSynchronize(); // wai for all kernels to finish so that the level array is updated
    timer.stop();
    double kernelTime = timer.elapsed();
    printf("\033[0;34m"); // set colour to blue
    printf("Kernel Time: %f ms\n", kernelTime);
    printf("\033[0m"); // reset color

    // check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        // in red
        printf("\033[0;31m");
        printf("CUDA error: %s\n", hipGetErrorString(error));
        // reset color
        printf("\033[0m");
    }

    // printf("Kernel executed successfully\n");

    // 6. Copy the result back to the host
    hipMemcpy(level, deviceLevel, vertices * sizeof(int), hipMemcpyDeviceToHost);

    // printf("Copied back to host successfully\n");

    // 7. Write the result to the output file
    for (int i = 0; i < vertices; i++)
    {
        fprintf(outputFile, "%d %d\n", i, level[i]);
    }

    // Close files
    fclose(inputFile);
    fclose(outputFile);

    // Free host memory
    free(srcPtrs);
    free(dst);
    free(level);

    // Free device memory
    timer.start();
    hipFree(deviceSrc);
    hipFree(deviceDst);
    hipFree(deviceLevel);
    hipFree(deviceVertexVisited);
    timer.stop();
    double deallocationTime = timer.elapsed();

    printf("Time taken to deallocate device memory: %f ms\n", deallocationTime);

    double totalGPUTime = allocationTime + copyingTime + kernelTime + deallocationTime;

    printf("\033[0;32m"); // set color to green
    printf("Total GPU time: %f ms\n", totalGPUTime);
    printf("\033[0m"); // reset color

    return 0;
}