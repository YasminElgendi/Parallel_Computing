#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include "graph.h"
#include "timer.h"

/*
 Vertex Centric Direction-Based Approach:
 - This is a hybrid implementation of the push and pull approaches
 - It starts with the top-down approach and then switches to the bottom-up approach
 - At first the bottom-up approach is inefficient since it has to check all the neighbours of the vertex and the vertex will be not any in the previous level
 - In the Initial iterations th etop-down approach is more efficient since there are very few neighbours in the previous level
 - To implement the hybrid approach the kernels will be called based on the current level
 - The cuda kernels will not be changed
 - Only the calling of the kernels in the main function will be changed
*/

__global__ void vertex_centric_pull_bfs(int *srcPtrs, int *dst, int *level, int currentLevel, int vertices, int edges, int *vertexVisited)
{
    int vertex = blockIdx.x * blockDim.x + threadIdx.x;

    if (vertex < vertices)
    {
        if (level[vertex] == -1)
        {
            // check if my neighbours are in the previous level
            int start = srcPtrs[vertex];
            int end = srcPtrs[vertex + 1];
            for (int i = start; i < end; i++)
            {
                int neighbour = dst[i];
                if (level[neighbour] == currentLevel - 1)
                {
                    level[vertex] = currentLevel;
                    *vertexVisited = 1;
                    break; // if one of the vertex neighbours has been visited then no need to check the rest of the neighbours (we're just setting the level of the current vertex)
                }
            }
        }
    }
}

__global__ void vertex_centric_push_bfs(int *srcPtrs, int *dst, int *level, int currentLevel, int vertices, int edges, int *vertexVisited)
{
    // each thread is assigned a vertex
    // since this is considered a 1-D array we will use the the x index to get the vertex for each thread
    // following the basis of a vector addition
    int vertex = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary conditions
    // check if the vertex is inside the graph
    if (vertex < vertices)
    {
        // printf("Vertex: %d, Vertex Visited: %d\n", vertex, *vertexVisited);
        if (level[vertex] == currentLevel - 1) // then loop on all neighbours of the vertex
        {
            // get the starting and ending index of the edges of the vertex
            // the srcPtrs contain the starting index of the edges of the vertex for each row (vertex)
            int start = srcPtrs[vertex];
            int end = srcPtrs[vertex + 1];

            // iterate over the neighbours of the vertex
            for (int i = start; i < end; i++)
            {
                int neighbour = dst[i];
                // printf("Vertex: %d, Neighbour: %d, Level: %d, Current Level: %d\n", vertex, neighbour, level[neighbour], currentLevel);
                // check if the neighbour has not been visited
                if (level[neighbour] == -1)
                {
                    level[neighbour] = currentLevel;
                    *vertexVisited = 1;
                }
            }
        }
    }
}

__host__ void cpu_bfs(int *srcPtrs, int *dst, int *level, int vertices, int edges)
{
}

int main(char argc, char *argv[])
{

    if (argc < 4)
    {
        printf("Please provide the paths of the input and output files and the source vertex\n");
        return 1;
    }

    FILE *inputFile;
    FILE *outputFile;
    int srcVertex = atoi(argv[3]);
    Timer timer;

    inputFile = fopen(argv[1], "r");
    outputFile = fopen(argv[2], "w");

    if (!inputFile || !outputFile)
    {
        printf("Please provide the correct path of both files");
        return 1;
    }

    // 1. Allocate host memory for the graph
    int vertices, edges;

    // Read the number of vertices and edges
    fscanf(inputFile, "%d %d", &vertices, &edges);

    int *srcPtrs = (int *)malloc((vertices + 1) * sizeof(int));
    int *dst = (int *)malloc(edges * sizeof(int));
    int *level = (int *)malloc(vertices * sizeof(int));
    // unsigned int *vertexVisited = (unsigned int *)malloc(sizeof(unsigned int));

    // printf("Host memory allocated successfully\n");

    // Initialize the level of each vertex to -1
    // and the source vertex to 0
    for (int i = 0; i < vertices; i++)
    {
        if (i == srcVertex)
            level[i] = 0;
        else
            level[i] = -1;
    }

    // Create a graph using the CSR representation

    // Construct the graph using the CSR representation
    CSRGraph(inputFile, srcPtrs, dst, edges);

    // 2. Allocate device memory for the graph
    timer.start();
    int *deviceSrc;
    int *deviceDst;
    int *deviceLevel;
    int *deviceVertexVisited;

    hipMalloc((void **)&deviceSrc, (vertices + 1) * sizeof(int));
    hipMalloc((void **)&deviceDst, edges * sizeof(int));
    hipMalloc((void **)&deviceLevel, vertices * sizeof(int));
    hipMalloc((void **)&deviceVertexVisited, sizeof(int));

    // printf("\nDevice memory allocated successfully\n");
    timer.stop();
    double allocationTime = timer.elapsed();
    printf("Time taken to allocate device memory: %f ms\n", allocationTime);

    // 3. Copy memory to the device
    timer.start();
    hipMemcpy(deviceSrc, srcPtrs, (vertices + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceDst, dst, edges * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceLevel, level, vertices * sizeof(int), hipMemcpyHostToDevice);
    timer.stop();

    // printf("Copied to GPU successfully\n");
    double copyingTime = timer.elapsed();

    printf("Time taken to copy memory to the device: %f ms\n", copyingTime);

    // 4. Set the number of threads and blocks
    int threadsPerBlock = 256;
    int blocksPerGrid = (vertices + threadsPerBlock - 1) / threadsPerBlock;
    unsigned vertexVisited = 1;

    int currentLevel = 1; // we start from level 1 since we already set the level of the source vertex to 0

    // printf("Launching Kernel\n");
    // 5. Launch the kernel
    timer.start();
    while (vertexVisited)
    {
        vertexVisited = 0;
        hipMemcpy(deviceVertexVisited, &vertexVisited, sizeof(unsigned int), hipMemcpyHostToDevice); // copy the vertexVisited to the device before the launch of each kernel

        // kernel processes each level
        // the kernel will be called for each level in the graph
        // global synchronisation across different levels
        if (currentLevel == 1)
            vertex_centric_push_bfs<<<threadsPerBlock, blocksPerGrid>>>(deviceSrc, deviceDst, deviceLevel, currentLevel, vertices, edges, deviceVertexVisited);
        else
            vertex_centric_pull_bfs<<<threadsPerBlock, blocksPerGrid>>>(deviceSrc, deviceDst, deviceLevel, currentLevel, vertices, edges, deviceVertexVisited);

        hipMemcpy(&vertexVisited, deviceVertexVisited, sizeof(unsigned int), hipMemcpyDeviceToHost); // copy the vertexVisited back to the host after the kernel has finished to check whether any vertex has been visited if not the max depth reached
        currentLevel++;
    }

    hipDeviceSynchronize(); // wai for all kernels to finish so that the level array is updated
    timer.stop();
    double kernelTime = timer.elapsed();
    printf("\033[0;34m"); // set colour to blue
    printf("Kernel Time: %f ms\n", kernelTime);
    printf("\033[0m"); // reset color

    // check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        // in red
        printf("\033[0;31m");
        printf("CUDA error: %s\n", hipGetErrorString(error));
        // reset color
        printf("\033[0m");
    }

    // printf("Kernel executed successfully\n");

    // 6. Copy the result back to the host
    hipMemcpy(level, deviceLevel, vertices * sizeof(int), hipMemcpyDeviceToHost);

    // printf("Copied back to host successfully\n");

    // 7. Write the result to the output file
    for (int i = 0; i < vertices; i++)
    {
        fprintf(outputFile, "%d %d\n", i, level[i]);
    }

    // Close files
    fclose(inputFile);
    fclose(outputFile);

    // Free host memory
    free(srcPtrs);
    free(dst);
    free(level);

    // Free device memory
    timer.start();
    hipFree(deviceSrc);
    hipFree(deviceDst);
    hipFree(deviceLevel);
    hipFree(deviceVertexVisited);
    timer.stop();
    double deallocationTime = timer.elapsed();

    printf("Time taken to deallocate device memory: %f ms\n", deallocationTime);

    double totalGPUTime = allocationTime + copyingTime + kernelTime + deallocationTime;

    printf("\033[0;32m"); // set color to green
    printf("Total GPU time: %f ms\n", totalGPUTime);
    printf("\033[0m"); // reset color

    return 0;
}