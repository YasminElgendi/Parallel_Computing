#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <fstream>

__global__ void kernel(float *d_array, float target_value, int size)
{
}

int getSize(FILE *input)
{
    int size = 1;
    char c;
    while ((c = fgetc(input)) != EOF)
    {
        if (c == '\n')
            size++;
    }
    rewind(input);
    return size;
}

void readInput(FILE *input, float *array, int size)
{
    for (int i = 0; i < size; i++)
    {
        fscanf(input, "%f", &array[i]);
    }
}

int main(char argc, char *argv[])
{
    if (argc < 3)
    {
        printf("Please provide the path of the input file and the target element\n");
        return 1;
    }

    FILE *inputFile;
    float target_value = atof(argv[2]);

    inputFile = fopen(argv[1], "r");

    if (inputFile == NULL)
    {
        printf("Error opening file\n");
        return 1;
    }

    int size = getSize(inputFile);

    printf("Size: %d\n", size);

    float *array, *result;

    float *d_array, *d_result;

    array = (float *)malloc(size * sizeof(float));
    result = (float *)malloc(2 * sizeof(float));

    readInput(inputFile, array, size);

    for (int i = 0; i < size; i++)
    {
        printf("%0.1f\n", array[i]);
    }

    // Allocate device memory
    hipMalloc((void **)&d_array, sizeof(float) * size);
    hipMalloc((void **)&d_result, sizeof(float) * 2);

    // 8. Deallocate device memory
    hipFree(d_array);
    hipFree(d_result);

    // 9. Deallocate host memory
    free(array);
    free(result);

    fclose(inputFile);

    return 0;
}
