#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#define STBI_MSC_SECURE_CRT
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include "./include/dirent.h"
#include "./include/stb/stb_image_write.h"
#include "read_data.h"

#define OUTPUT_TILE_WIDTH 16 // => 16 x 16 = 256

__constant__ float constant_mask[MAX_MASK_SIZE * MAX_MASK_SIZE]; // constant memory for the mask

// 3. kernel3: tiling where each block matches the output tile size.
// The size of the block matches the size of the input tile
__global__ void kernel3_batch(unsigned char *output_images, float *input_images, int width, int height, int comp, int mask_size, int batch_size, int input_tile_width)
{
    // get the pixel index in the output image
    int out_column = blockIdx.x * OUTPUT_TILE_WIDTH + threadIdx.x;
    int out_row = blockIdx.y * OUTPUT_TILE_WIDTH + threadIdx.y;
    int depth = blockIdx.z * blockDim.z + threadIdx.z;

    // indices to access the shared memory
    int in_column = out_column - mask_size / 2;
    int in_row = out_row - mask_size / 2;

    // Indices that are used to access the input image
    int start_block_column = blockIdx.x * OUTPUT_TILE_WIDTH - mask_size / 2;
    int start_block_row = blockIdx.y * OUTPUT_TILE_WIDTH - mask_size / 2;

    // the number of elements that each thread should load
    int elements_per_thread = ((input_tile_width * input_tile_width) / (OUTPUT_TILE_WIDTH * OUTPUT_TILE_WIDTH)) + 1;

    // STEPS:
    // 1. Load data into shared memory
    // some threads will load more than one pixel to the shared memeory
    // each thread will load 3 values corresponding to the three channels
    extern __shared__ float shared_input[]; // Example: Allocate the input tile size of shared memory

    if (depth < batch_size)
    {
        int stride = OUTPUT_TILE_WIDTH * OUTPUT_TILE_WIDTH;

        for (int i = 0; i < elements_per_thread; i++) // for each thread iterate ove the elements that it should load => same element in eaxg row in the inpyt tile
        {
            // the thread index with respect to the rest of the threads
            // if the output tile is 3x3 => 9 threads
            // the thread_index should go from 0-8
            int thread_index = threadIdx.y * OUTPUT_TILE_WIDTH + threadIdx.x;

            // since a single thread loads more than one input element
            // the step is the difference between elements for which a single thread loads its in
            int thread_index_step = thread_index + (i * stride);

            // get the indices of the the thread with respect to the input tile
            // if a thread is number 8 and the input tile is 5x5 => then the thread loads into the cell 1,3 in the shared memory
            int shm_index_row = thread_index_step / input_tile_width;
            int shm_index_col = thread_index_step - (shm_index_row * input_tile_width);

            if (shm_index_col >= 0 && shm_index_col < input_tile_width && shm_index_row >= 0 && shm_index_row < input_tile_width)
            {

                // get the index of the thread with respect to the input image
                // use the in_column and in_row => wrong
                // use the shared memory indices
                int input_index_col = start_block_column + shm_index_col;
                int input_index_row = start_block_row + shm_index_row;

                if (input_index_col >= 0 && input_index_col < width && input_index_row >= 0 && input_index_row < height)
                {
                    for (int c = 0; c < comp; c++) // this does not support memory coalescing
                    {
                        // Load the pixel value from the input image into the shared memory if the image is in bounds
                        shared_input[(shm_index_row * input_tile_width + shm_index_col) * comp + c] = input_images[(width * height * depth + input_index_row * width + input_index_col) * comp + c];
                    }
                }
                else
                {
                    // this does not support memory coalescing
                    // since the image is saved to memory where each pixel is saved in three consecutive cells
                    // we want the threads to load consecutive cells
                    // if I have 3 threads and three pixels => thread 0 loads first channel of each pixel, thread 1 loads the second channel of each pixel, thread 2 loads the third channel of each pixel => ezay ba2a
                    for (int c = 0; c < comp; c++)
                    {
                        // Insert a 0 if the index is out of bounds => ghost cells
                        shared_input[(shm_index_row * input_tile_width + shm_index_col) * comp + c] = 0.0f;
                    }
                }
            }
        }
    }

    __syncthreads();

    // 2. Compute the output tile

    // all threads will participate in computing the pixel value
    // no need to check if the thread is within the boundaries
    if (out_column < width && out_row < height && depth < batch_size)
    {
        float pixel_value = 0.0f;
        for (int c = 0; c < comp; c++)
        {
            // iterate over the mask elements => surrounding box
            for (int mask_row = 0; mask_row < mask_size; mask_row++) // rows
            {
                for (int mask_column = 0; mask_column < mask_size; mask_column++) // columns
                {
                    pixel_value += shared_input[((threadIdx.y + mask_row) * input_tile_width + (threadIdx.x + mask_column)) * comp + c] * constant_mask[mask_row * mask_size + mask_column];
                }
            }
        }

        pixel_value = fminf(fmaxf(pixel_value, 0.0f), 1.0f); // clamp the pixel value to be in the range [0, 1]

        pixel_value = pixel_value * 255; // scale the pixel value to be in the range [0, 255]

        // 3. Write the output tile to the output image
        output_images[width * height * depth + out_row * width + out_column] = (unsigned char)pixel_value;
    }
}

// Dealing with the output images
// Deals with the threads, block and grid dimensions
// Calls the kernel to calculate the output images
// Transfers the output images from device to host
// Saves the output images
void calculateOutput(int width, int height, int channels, int depth, unsigned char *output_images, unsigned char *device_outputs, float *device_images, int mask_size, char *output_folder_path, char **output_image_filenames)
{
    // calculate the block and grid size

    int input_tile_width = OUTPUT_TILE_WIDTH + mask_size - 1;

    dim3 block_dim(OUTPUT_TILE_WIDTH, OUTPUT_TILE_WIDTH);
    int grid_columns = ceil((float)width / OUTPUT_TILE_WIDTH);
    int grid_rows = ceil((float)height / OUTPUT_TILE_WIDTH);
    dim3 grid_dim(grid_columns, grid_rows, depth);

    int shared_memory_size = input_tile_width * input_tile_width * channels * sizeof(float);

    // call the kernel on the batch of images read
    kernel3_batch<<<grid_dim, block_dim, shared_memory_size>>>(device_outputs, device_images, width, height, channels, mask_size, depth, input_tile_width);

    // transfer the output images from device to host
    hipMemcpy(output_images, device_outputs, width * height * depth * sizeof(unsigned char), hipMemcpyDeviceToHost);

    printf("OUTPUT IMAGES COPIED TO HOST\n");

    // Get full output path
    char full_output_path[256];

    // save images
    for (int i = 0; i < depth; i++)
    {
        sprintf(full_output_path, "%s/%s", output_folder_path, output_image_filenames[i]);
        printf("FULL OUTPUT PATH: %s\n", full_output_path);

        stbi_write_jpg(full_output_path, width, height, 1, output_images + i * width * height, 100);
    }
}

int main(char argc, char *argv[])
{
    // Read and check command line arguments
    char *input_folder_path;
    char *output_folder_path;
    char *mask_file_path;

    int batch_size = readCommandLineArguments(argc, argv, &input_folder_path, &output_folder_path, &mask_file_path);

    // Allocate memory for filenames to save output images
    char **output_image_filenames = (char **)malloc(batch_size * sizeof(char *)); // Dynamic allocation for image names

    printf("%s\n", input_folder_path);
    printf("%s\n", output_folder_path);

    // First get the dimension of the images => all images are supposed to have the same dimension => get the dimension of the first image
    int WIDTH, HEIGHT, CHANNELS; // image dimensions for all images in the input folder
    bool success = getImageDimensions(input_folder_path, &WIDTH, &HEIGHT, &CHANNELS);

    if (!success)
    {
        printf("Error: failed to read image dimensions\n");
        exit(1);
    }

    // 1. ALlocate host and device memory based on the batch size

    // 1.1 Host memory
    unsigned char *output_images = (unsigned char *)malloc(batch_size * WIDTH * HEIGHT * sizeof(unsigned char));

    // 1.2 Device memory
    float *device_images;
    unsigned char *device_outputs;

    hipMalloc((void **)&device_images, sizeof(float) * WIDTH * HEIGHT * CHANNELS * batch_size);
    hipMalloc((void **)&device_outputs, sizeof(unsigned char) * WIDTH * HEIGHT * batch_size);

    // 2. Read the mask and copy it to the contstant memory
    FILE *mask_file = fopen(mask_file_path, "r");
    if (!mask_file)
    {
        printf("Error: failed to read mask\n");
        exit(1);
    }

    // Read mask from file => same mask applied to all channels

    int mask_size = readMaskSize(mask_file);                              // read mask size
    float *mask = (float *)malloc(mask_size * mask_size * sizeof(float)); // allocate memory for the mask
    readMask(mask_file, mask, mask_size);                                 // read mask elements
    printMask(mask, mask_size);                                           // print mask elements

    // Copy Filter to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(constant_mask), mask, mask_size * mask_size * sizeof(float));

    // 3. Read images as batches from the input folder
    DIR *input_directory;
    struct dirent *entry;
    int batch_count = 0;

    if ((input_directory = opendir(input_folder_path)) != NULL)
    {
        printf("Input directory opened\n");

        // Read all images in batches and send to the GPU for convolution
        while ((entry = readdir(input_directory)) != NULL)
        {
            if (entry->d_type == DT_REG)
            {
                const char *image_name = entry->d_name;
                printf("Image File name = %s\n", image_name);
                output_image_filenames[batch_count] = strdup(image_name);

                // Get full input path
                char full_input_path[256];
                sprintf(full_input_path, "%s/%s", input_folder_path, image_name);
                printf("FULL INPUT PATH: %s\n", full_input_path);

                // Read the image
                int width, height, channels;
                unsigned char *input_image = readImage(full_input_path, &width, &height, &channels);
                printf("width = %d, height = %d, channels = %d\n", width, height, channels);
                if (input_image == NULL)
                {
                    printf("Error: failed to read image\n");
                    exit(1);
                }

                assert(width == WIDTH && height == HEIGHT && channels == CHANNELS);

                // normalize image => convert to float between 0 and 1
                float *normalized_image = (float *)malloc(WIDTH * HEIGHT * CHANNELS * sizeof(float));
                for (size_t i = 0; i < WIDTH * HEIGHT * CHANNELS; i++)
                {
                    normalized_image[i] = (float)input_image[i] / 255.0f;
                }

                // Copy image data from host to device
                hipMemcpy(device_images + batch_count * WIDTH * HEIGHT * CHANNELS, normalized_image, WIDTH * HEIGHT * CHANNELS * sizeof(float), hipMemcpyHostToDevice);

                printf("IMAGE COPIED TO GPU\n");

                free(input_image);
                free(normalized_image);

                batch_count++;
                if (batch_count >= batch_size)
                {
                    printf("BATCH COUNT: %d\n", batch_count);
                    calculateOutput(WIDTH, HEIGHT, CHANNELS, batch_size, output_images, device_outputs, device_images, mask_size, output_folder_path, output_image_filenames);
                    batch_count = 0;
                }
            }
        }

        printf("BATCH COUNT: %d\n", batch_count);
        if (batch_count != 0) // if the file_count % batch_size != 0
        {
            calculateOutput(WIDTH, HEIGHT, CHANNELS, batch_count, output_images, device_outputs, device_images, mask_size, output_folder_path, output_image_filenames);
        }

        // free dynamically allocated host memory
        free(mask);
        free(output_images);
        free(output_image_filenames);

        // free device memory
        hipFree(device_images);
        hipFree(device_outputs);

        // close the opened folders and files
        fclose(mask_file);
        closedir(input_directory);
    }
    else
    {
        // Error opening input directory
        perror("");
        return EXIT_FAILURE;
    }

    return 0;
}