#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <fstream>

__global__ void kernel(float *d_array, float target_value, int size, float *result)
{
    const int num_of_threads = blockDim.x * gridDim.x;
	const int index = blockIdx.x * blockDim.x + threadIdx.x;
}

int getSize(FILE *input)
{
    int size = 1;
    char c;
    while ((c = fgetc(input)) != EOF)
    {
        if (c == '\n')
            size++;
    }
    rewind(input);
    return size;
}

void readInput(FILE *input, float *array, int size)
{
    for (int i = 0; i < size; i++)
    {
        fscanf(input, "%f", &array[i]);
    }
}

int main(char argc, char *argv[])
{
    if (argc < 3)
    {
        printf("Please provide the path of the input file and the target element\n");
        return 1;
    }

    FILE *inputFile;
    float target_value = atof(argv[2]);

    inputFile = fopen(argv[1], "r");

    if (inputFile == NULL)
    {
        printf("Error opening file\n");
        return 1;
    }

    int size = getSize(inputFile);

    printf("Size: %d\n", size);

    float *array, *result;

    float *d_array, *d_result;

    array = (float *)malloc(size * sizeof(float));
    result = (float *)malloc(2 * sizeof(float));

    readInput(inputFile, array, size);

    for (int i = 0; i < size; i++)
    {
        printf("%0.1f\n", array[i]);
    }

    // Allocate device memory
    hipMalloc((void **)&d_array, sizeof(float) * size);
    hipMalloc((void **)&d_result, sizeof(float) * 2);

    // Transfer data from host to device memory
    hipMemcpy(d_array, array, sizeof(float) * size, hipMemcpyHostToDevice);

    int block_size = 256;
    int grid_size = (size + block_size - 1) / block_size;

    kernel<<<grid_size, block_size>>>(d_array, size, target_value, d_result);

    // Transfer data back to host memory
    hipMemcpy(result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    // 8. Deallocate device memory
    hipFree(d_array);
    hipFree(d_result);

    // 9. Deallocate host memory
    free(array);
    free(result);

    fclose(inputFile);

    return 0;
}
